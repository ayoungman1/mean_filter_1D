#include "hip/hip_runtime.h"
//��ֵ�˲�����Ϊboxfilter


#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "dehaze_kernel.h"
#define TILE_DIM 16
#define BLOCKSIZE 128



__global__ void d_boxfilter_x_global(float *src, float *dst, int width, int height, int r)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int offset = 1;
	int num = (width + 2 * r + 2 * BLOCKSIZE - 1) / (2 * BLOCKSIZE);		//ÿһ���߳̿鱻BLOCKSIZE*2�ָ����num��segment
	int len = num * 2 * BLOCKSIZE;
	int extra = len - r - width;
	float scale = 1.0f / (float)((r << 1) + 1);

	__shared__ float sum[35]; sum[0] = 0;

	extern __shared__ float temp[];

	if (bid < height)
	{
		//�߽����
		for (int i = tid; i < r; i += BLOCKSIZE)
		{
			temp[i] = src[bid*width + 0];								//ǰr�������һ�еĵ�һ��Ԫ��
		}
		//__syncthreads();

		for (int i = tid; i < width; i += BLOCKSIZE)
		{
			temp[r + i] = src[bid * width + i];
		}
		//__syncthreads();

		for (int i = tid; i < extra; i += BLOCKSIZE)
		{
			temp[r + width + i] = src[(bid + 1) * width - 1];			//���extra�������һ�����һ��Ԫ��
		}
		__syncthreads();


		for (int cnt = 0; cnt < num; ++cnt)								//numΪ��������
		{
			int bias = cnt * BLOCKSIZE * 2;
			//__syncthreads();
			//up-sweep phase
			for (int j = BLOCKSIZE; j > 0; j >>= 1)
			{
				if (tid < j)
				{
					int ai = bias + offset * (2 * tid + 1) - 1;
					int bi = bias + offset * (2 * tid + 2) - 1;
					temp[bi] += temp[ai];
				}
				offset *= 2;
				__syncthreads();
			}
			//down-sweep phase		
			if (tid == 0)
			{
				sum[cnt + 1] = temp[(cnt + 1) * BLOCKSIZE * 2 - 1] + sum[cnt]; //֮��ÿ�е�ÿ��segment[i]Ҫ����sum[i]������sum[i]��ʾǰi�����ݿ�������Ԫ�صĺͣ�ÿ�����ݿ��� BLOCKSIZE * 2��С
				temp[(cnt + 1) * BLOCKSIZE * 2 - 1] = 0;
			}
			__syncthreads();
			for (int j = 1; j < (BLOCKSIZE * 2); j *= 2)
			{
				offset >>= 1;
				if (tid < j)
				{
					int ai = bias + offset * (2 * tid + 1) - 1;
					int bi = bias + offset * (2 * tid + 2) - 1;

					float t = temp[ai];
					temp[ai] = temp[bi];
					temp[bi] += t;
				}
				__syncthreads();
			}
		}
		for (int i = tid; i < width; i += BLOCKSIZE)
		{
			float sum_box = temp[i + 2 * r + 1] + sum[(i + 2 * r + 1) / (BLOCKSIZE * 2)] - temp[i] - sum[i / (BLOCKSIZE * 2)];		//sumֻ�ǵ�i + 2 * r + 1֮ǰ������Ԫ��֮�Ͳ�������i + 2 * r + 1��Ԫ��
			dst[bid * width + i] = sum_box * scale;
			//dst[bid * width + i] = temp[i];
			//dst[bid * width + i] = src[bid * width + i];
		}
	}
}


//2018.12.18,����HILLIS����ͼ
//sipenghui
__global__ void d_boxfilter_x_hillis(float *src, float *dst, int width, int height, int r)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int offset = 1;
	int num = (width + 2 * r + 2 * BLOCKSIZE - 1) / (2 * BLOCKSIZE);		//ÿһ���߳̿鱻BLOCKSIZE*2�ָ����num��segment
	int len = num * 2 * BLOCKSIZE;
	int extra = len - r - width;
	float scale = 1.0f / (float)((r << 1) + 1);

	__shared__ float sum[35]; sum[0] = 0;

	extern __shared__ float temp[];

	if (bid < height)
	{
		//�߽����
		for (int i = tid; i < r; i += 2*BLOCKSIZE)						//2019.2.28�޸�CLOCKSIZE��2
		{
			temp[i] = src[bid*width + 0];								//ǰr�������һ�еĵ�һ��Ԫ��
			temp[len*2 + i] = src[bid*width + 0];

		}

		for (int i = tid; i < width; i += 2*BLOCKSIZE)
		{
			temp[r + i] = src[bid * width + i];
			temp[len * 2 + r + i] = src[bid * width + i];

		}

		for (int i = tid; i < extra; i += 2*BLOCKSIZE)
		{
			temp[r + width + i] = src[(bid + 1) * width - 1];			//���extra�������һ�����һ��Ԫ��
			temp[len * 2 + r + width + i] = src[(bid + 1) * width - 1];

		}
		__syncthreads();

		int pout = 0;
		int pin = 1;

		for (int cnt = 0; cnt < num; ++cnt)								//numΪ��������
		{
			int bias = cnt * BLOCKSIZE * 2;

			pout = 0;
			pin = 1;

			for (offset = 1; offset < BLOCKSIZE * 2; offset *= 2)
			{
				pout = 1 - pout;
				pin = 1 - pin;

				//if (tid<2*BLOCKSIZE)
				//	temp[len*2+bias + tid] = temp[bias + tid];
				//__syncthreads();
				//temp[pout*len + bias + tid] = temp[pin*len + bias + tid];		//�汾1

				int ai = pout*len + bias + tid;
				int bi = pin*len + bias + tid;
				if (tid >= offset && tid < 2 * BLOCKSIZE)						//�汾2
					//temp[bias + tid] += temp[bias + tid - offset];	//2019.2.28�˴����ڳ�ͻ
					temp[ai] = temp[bi] + temp[bi - offset];
				else
					temp[ai] = temp[bi];

				__syncthreads();
			}

			if (tid == 2 * BLOCKSIZE-1)
			{
				sum[cnt + 1] = temp[pout*len + bias + tid] + sum[cnt];
			}
			if (tid < 2 * BLOCKSIZE)
				temp[pout*len + bias + tid] = temp[pout*len + bias + tid] - temp[len * 2 + bias + tid];

			__syncthreads();
		}


		for (int i = tid; i < width; i += BLOCKSIZE)
		{
			float sum_box = temp[pout*len + i + 2 * r + 1] + sum[(i + 2 * r + 1) / (BLOCKSIZE * 2)] - temp[pout*len + i] - sum[i / (BLOCKSIZE * 2)];		//sumֻ�ǵ�i + 2 * r + 1֮ǰ������Ԫ��֮�Ͳ�������i + 2 * r + 1��Ԫ��
			dst[bid * width + i] = sum_box * scale;
		}
	}
}


extern "C"
void boxfilter(float *id, float *od, float *d_temp, float *d_temp1, int width, int height, int r)
{

	int num_shared1 = ((width + 2 * r + BLOCKSIZE * 2 - 1) / (BLOCKSIZE * 2)) * 2 * BLOCKSIZE;
	int num_shared2 = ((height + 2 * r + BLOCKSIZE * 2 - 1) / (BLOCKSIZE * 2)) * 2 * BLOCKSIZE;

	dim3 grid1(width / TILE_DIM + 1, height / TILE_DIM + 1);
	dim3 grid2(height / TILE_DIM + 1, width / TILE_DIM + 1);
	dim3 block(TILE_DIM, TILE_DIM);



	//d_boxfilter_x_global << < height, BLOCKSIZE, num_shared1 * sizeof(float) >> > (id, d_temp, width, height, r);
	//transpose << <grid1, block >> >(d_temp1, d_temp, width, height);
	//d_boxfilter_x_global << < width, BLOCKSIZE, num_shared2 * sizeof(float) >> > (d_temp1, od, height, width, r);

	d_boxfilter_x_hillis << < height, BLOCKSIZE * 2, num_shared1 * sizeof(float)*3 >> > (id, d_temp, width, height, r);
	transpose << <grid1, block >> >(d_temp1, d_temp, width, height);
	d_boxfilter_x_hillis << < width, BLOCKSIZE * 2, num_shared2 * sizeof(float)*3 >> > (d_temp1, od, height, width, r);


}
